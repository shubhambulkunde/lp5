#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;

// VectorAdd parallel function
__global__ void vectorAdd(int *a, int *b, int *result, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n)
    {
        result[tid] = a[tid] + b[tid];
    }
}

int main()
{
    int *a, *b, *c;
    int *a_dev, *b_dev, *c_dev;
    int n = 1 << 24; // Total number of elements

    a = new int[n];
    b = new int[n];
    c = new int[n];

    int *d = new int[n]; // For serial addition
    int size = n * sizeof(int);

    // Allocate memory on device
    hipMalloc(&a_dev, size);
    hipMalloc(&b_dev, size);
    hipMalloc(&c_dev, size);

    // Array initialization
    for (int i = 0; i < n; i++)
    {
        a[i] = 1;
        b[i] = 2;
        d[i] = a[i] + b[i]; // calculating serial addition
    }

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // Copy data from host to device
    hipMemcpy(a_dev, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice);

    int threads = 1024;
    int blocks = (n + threads - 1) / threads;

    hipEventRecord(start);

    // Parallel addition kernel invocation
    vectorAdd<<<blocks, threads>>>(a_dev, b_dev, c_dev, n);

    hipEventRecord(end);
    hipEventSynchronize(end);

    float time = 0.0;
    hipEventElapsedTime(&time, start, end);

    // Copy result back to host
    hipMemcpy(c, c_dev, size, hipMemcpyDeviceToHost);

    // Calculate the error term.
    int error = 0;
    for (int i = 0; i < n; i++)
    {
        error += d[i] - c[i];
    }

    cout << "Error : " << error << endl;
    cout << "Time Elapsed:  " << time << " milliseconds" << endl;

    // Free memory
    delete[] a;
    delete[] b;
    delete[] c;
    delete[] d;
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);

    return 0;
}


//output

// Error : 0
// Time Elapsed:  2.34865 milliseconds
